/*
 * cuda_checkout.cpp
 *
 *  Created on: Oct 26, 2020
 *      Author: bflynt
 */



#include <hip/hip_runtime.h>
#include <iostream>



int main() {

	// CUDA Always returns >0 for Device Count
	// It will emulate on the CPU the device if no device exists
	int count = -999;
	hipGetDeviceCount(&count);
	std::cout << "hipGetDeviceCount = " << count << std::endl;
	
	// CUDA Always returns a 0 for initial device
	// even when no device exists since it will emulate on the CPU the device
	int initial = 0;
	std::cout << " = " << initial << std::endl;
	
	int device = -999;
	hipGetDevice(&device);
	std::cout << "hipGetDevice = " << device << std::endl;
	
	

	return 0;
}





